#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stencil.cuh"
#include "profile.cuh"

#include <iostream>
#include <string> 
#include <random>

using std::cout;
using std::endl;

int main(int argc, char *argv[])
{
    // if (argc != 3)
    // {
    //     cout << "Usage ./task1 n R threads per block" << endl;
    //     return 0;
    // }
    size_t N = 4; //std::stoi(argv[1]);
    size_t R = 2; 
    size_t threads_per_block = 1024;

    float *image, *mask, *output;     
    float *d_image, *d_mask, *d_output;
    size_t image_size = N * N * sizeof(float);
    size_t mask_size = (2 * R + 1) * sizeof(float);
    

    hipError_t cudaStatus;
    // Generate Random Values for kernel
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source()); 
    std::uniform_real_distribution<float> dist(-1.0,1.0);
    
    // Allocate space for device and host array a
    hipMalloc((void **)&d_image, image_size);
    hipMalloc((void **)&d_mask, mask_size);
    hipMalloc((void **)&d_output, image_size);
    
    image = (float *)malloc(image_size);
    mask = (float *)malloc(mask_size);
    output = (float *)malloc(image_size);
    
    // Fill a, b, c array on host
    for(size_t i = 0; i < N * N ; i++)
    {
        image[i] = dist(generator);
        output[i] = 0;
    }
    for(size_t i = 0; i < (2 * R + 1) ; i++)
    {
        mask[i] = dist(generator);
    }


    //Copy data from host to device
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, mask_size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, image_size, hipMemcpyHostToDevice);
    float time_taken = 0;
    {
        UnitGPUTime g;
        stencil(d_image , d_mask , d_output, N , R , threads_per_block);
        time_taken = g.getTime();
    }
    // Copy result back to host
    cudaStatus = hipMemcpy(output, d_output, image_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy returned error code %d after copying from kernel!\n", cudaStatus);
        return 0;
    }

    cout << output[N * N - 1] << endl << time_taken << endl;

    // Cleanup
    free(image);
    free(mask);
    free(output);
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
    return 0;
}