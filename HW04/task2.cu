#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stencil.cuh"
#include "profile.cuh"

#include <iostream>
#include <string> 
#include <random>

using std::cout;
using std::endl;

void cpu_stencil(const float* image ,const float* mask , float* out_cpu, int N, int R)
{
    float img_val;
    int pos_R = R;
    int neg_R = pos_R * -1;
    for(int i = 0 ; i < N ; i++)
    {
        out_cpu[i] = 0;
        for(int j = neg_R ; j <= pos_R ; j++ )
        {
            int i_j = i + j;
            if( (0 <= i_j) && (i_j < N) )
                img_val = image[i_j];
            else
                img_val = 1.0;
            out_cpu[i] += img_val * mask[j + R];
            
        }
    }
}
int main(int argc, char *argv[])
{
    if (argc != 4)
    {
        cout << "Usage ./task1 n R threads per block" << endl;
        return 0;
    }
    size_t N = std::stoi(argv[1]);
    size_t R = std::stoi(argv[2]);
    size_t threads_per_block = std::stoi(argv[3]);

    float *image, *mask, *output;     
    float *d_image, *d_mask, *d_output;
    size_t image_size = N * sizeof(float);
    size_t mask_size = (2 * R + 1) * sizeof(float);
    

    hipError_t cudaStatus;
    // Generate Random Values for kernel
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source()); 
    std::uniform_real_distribution<float> dist(-1.0,1.0);
    
    // Allocate space for device and host array a
    hipMalloc((void **)&d_image, image_size);
    hipMalloc((void **)&d_mask, mask_size);
    hipMalloc((void **)&d_output, image_size);
    
    image = (float *)malloc(image_size);
    mask = (float *)malloc(mask_size);
    output = (float *)malloc(image_size);
    //float* out_cpu = (float *)malloc(image_size);
    
    // Fill a, b, c array on host
    for(size_t i = 0; i < N ; i++)
    {
        image[i] = dist(generator);
        output[i] = 0;
    }
    for(size_t i = 0; i < (2 * R + 1) ; i++)
    {
        mask[i] = dist(generator);
    }

    //Copy data from host to device
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, mask_size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, image_size, hipMemcpyHostToDevice);
    float time_taken = 0;
    {
        UnitGPUTime g;
        stencil(d_image , d_mask , d_output, N , R , threads_per_block);
        time_taken = g.getTime();
    }
    // Copy result back to host
    cudaStatus = hipMemcpy(output, d_output, image_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy returned error code %d after copying from kernel!\n", cudaStatus);
        return 0;
    }

    cout << output[N - 1] << "," << std::log2(N) << "," << time_taken << endl;

    // cpu_stencil(image , mask ,out_cpu, N,  R);
    // for(size_t i = 0 ; i < N ; i++)
    // {
    //     if( abs( out_cpu[i] - output[i] ) > 1e-5 )
    //     {
    //         cout << "Diff at " << i << " C:" <<  out_cpu[i] << " G:" <<  output[i] << endl;
    //         //break;
    //     }
    // }

    // Cleanup
    free(image);
    free(mask);
    free(output);
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
    return 0;
}