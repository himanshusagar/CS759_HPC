#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "matmul.cuh"

#include <iostream>
#include <string> 
#include <random>

using std::cout;
using std::endl;

int main(void)
{
    int N = 10;
    int threads_per_block = 512;
    float *a, *b, *c;     
    float *a_d, *b_d, *c_d;
    int size = N * N * sizeof(float);
    hipError_t cudaStatus;
    // Generate Random Values for kernel
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source()); 
    std::uniform_int_distribution<int> dist(1,1000);
    
    // Allocate space for device and host array a
    hipMalloc((void **)&a_d, size);
    hipMalloc((void **)&b_d, size);
    hipMalloc((void **)&c_d, size);
    
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);
    
    // Fill a, b, c array on host
    for(int i = 0; i < N * N ; i++)
    {
        a[i] = i;
        b[i] = 1;
        c[i] = 0;
    }

    //Copy data from host to device
    hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);
    hipMemcpy(c_d, c, size, hipMemcpyHostToDevice);
    
    matmul(a_d , b_d , c_d, N , threads_per_block);
    // Copy result back to host
    cudaStatus = hipMemcpy(c, c_d, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy returned error code %d after copying from kernel!\n", cudaStatus);
        return 0;
    }
    //Priting out device filled output
    for(int i = 0; i < N ; i++)
    {
        for(int j = 0; j < N ; j++)
        {
            cout << c[i * N + j] << " ";
        }
        cout << endl;
    }

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}