
#include "hip/hip_runtime.h"



//vscale kernel to compute element-wise product between two vectors - a and b. Store result in b.
__global__ void vscale(const float *a, float *b, unsigned int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(0 <= index && index < n)
        b[index] = a[index] * b[index];
}