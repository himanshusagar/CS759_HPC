
#include "hip/hip_runtime.h"



__global__ void vscale(const float *a, float *b, unsigned int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(0 <= index && index < n)
        b[index] = a[index] * b[index];
}