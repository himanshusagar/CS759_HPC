#include "hip/hip_runtime.h"


#include <iostream>

using std::cout;
using std::endl;

__global__ void add(int *a, int *c)
{
    c[threadIdx.x] = 1;
    for( int i = 1 ; i <= a[threadIdx.x]; i++)
    {
        c[threadIdx.x] = c[threadIdx.x] * (i);
    }
    std::printf("%d!=%d\n", a[threadIdx.x], c[threadIdx.x] );
}

int main(void)
{
    int N = 16;
    int *a,  *c;       
    int *d_a,  *d_c;   
    int size = N * sizeof(int);
    hipError_t cudaStatus;
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_c, size);
    // Allocate space for host copies of a, b, c and setup input values
    a = (int *)malloc(size);
    c = (int *)malloc(size);

    // Fill A array on host
    for(int i = 0; i < N ; i++)
        a[i] = i+1;
    //Copy data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU with 1 block and N threads.
    add<<<1, N>>>(d_a, d_c);
    // Synchronize and see if we were successful.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return 0;
    }

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
    free(a);
    free(c);
    hipFree(d_a);
    hipFree(d_c);
    return 0;
}