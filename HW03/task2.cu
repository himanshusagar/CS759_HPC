#include "hip/hip_runtime.h"


#include <iostream>

using std::cout;
using std::endl;

__global__ void add(int *dA)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //dA[index] = 1;
    int sol = threadIdx.x + blockIdx.x;
    std::printf("%d %d" , index, sol);
}


int main(void)
{
    int N = 16;
    int *hA;     
    int *dA;
    int size = N * sizeof(int);
    hipError_t cudaStatus;
    // Allocate space for device and host array a
    hipMalloc((void **)&dA, size);
    hA = (int *)malloc(size);
    // Fill hA array on host
    for(int i = 0; i < N ; i++)
        hA[i] = 0;
    //Copy data from host to device
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU with 2 block and 8 threads.
    add<<<2, 8>>>(dA );
   // Copy result back to host
    cudaStatus = hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy returned error code %d after copying from kernel!\n", cudaStatus);
        return 0;
    }

    for(int i=0; i < N ; i++)
    {
        cout << hA[i] << " ";
    }
    cout << endl;
    // Cleanup
    free(hA);
    hipFree(dA);
    return 0;
}