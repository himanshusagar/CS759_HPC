#include "matmul.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "profile.cuh"

#include <iostream>
#include <string>
#include <random>

using std::cout;
using std::endl;

template <typename T>
int perf_matmul(int N, int threads_per_block)
{

  T *A, *B, *C;
  size_t size = N * N * sizeof(T);
  hipError_t cudaStatus;
  // Generate Random Values for kernel
  std::random_device entropy_source;
  std::mt19937 generator(entropy_source());
  std::uniform_real_distribution<T> dist(-1.0, 1.0);

  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&A, size);
  hipMallocManaged(&B, size);
  hipMallocManaged(&C, size);

  // initialize A,B and C matrices on the host
  for (int i = 0; i < N * N; i++)
  {
    A[i] = 1;
    B[i] = 2;
    C[i] = 0;
  }

  matmul<T>(A, B, C, threads_per_block);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Free memory
  hipFree(A);
  hipFree(B);
  return 0;
}

int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    cout << "Usage ./task1 N block_dim" << endl;
    return 0;
  }
  size_t N = std::stoi(argv[1]);
  size_t threads_per_block = std::stoi(argv[2]);
  perf_matmul<float>(N, threads_per_block);
  
}
