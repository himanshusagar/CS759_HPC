#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include "profile.cuh"

template <typename T>
__global__ void matmul_kernel(const T *A, const T *B, T *C, unsigned int N, unsigned int BLOCK_SIZE)
{
    extern __shared__ char smem[];
    T* sData = reinterpret_cast<T *>(smem);

    T* As = sData;
    T* Bs = sData + BLOCK_SIZE * BLOCK_SIZE;

    int tx = threadIdx.x; 
    int ty = threadIdx.y;

    // Global row and col index for bounds checking.
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;
    T cSol = 0;
    int limit = (N - 1)/BLOCK_SIZE + 1;

    for (int k = 0; k < limit; ++k) 
    {
        // See out of tile for As
        if(row < N && ( (k * BLOCK_SIZE + tx) < N)  )
            As[ty * BLOCK_SIZE + tx] = A[ row * N + k * BLOCK_SIZE + tx];
        else 
            As[ty * BLOCK_SIZE + tx] = 0;

        // See out of tile for As
        if(col < N && ( (k * BLOCK_SIZE + ty) < N ) )
            Bs[ty * BLOCK_SIZE + tx] = B[( k * BLOCK_SIZE + ty ) * N + col]; 
        else
            Bs[ty * BLOCK_SIZE + tx] = 0;
        
        __syncthreads();
        
        // We still need to check here because otherwise As and Bs would not be filled correctly.
        // But we only need to compute cSol for indices in range.
        if(row < N && col < N)
        for (int i = 0; i < BLOCK_SIZE; ++i)
        {  
            cSol += As[ty * BLOCK_SIZE + i] * Bs[i * BLOCK_SIZE + tx];
        }
        __syncthreads();
    }

    // Now fill back results for cSol's computed.
    if(row < N && col < N)
        C[row * N + col] = cSol;
}


template <typename T>
__host__ void matmul(const T *A, const T *B, T *C, unsigned int N,  unsigned int block_dim)
{
    // Launch simple kernel on GPU with 2 block and 8 threads.
    float f_N = N;
    float f_block_dim = block_dim;
    size_t grid_size = ceil(f_N / f_block_dim);
    dim3 dimBlock( block_dim, block_dim );
    dim3 dimGrid( grid_size , grid_size );
    size_t shared_mem_size = 2 * block_dim * block_dim * sizeof(T); // two mini matrices of size block_dim.
    //std::cout << block_dim << "X" << grid_size << " SM: " << shared_mem_size << " " << N << std::endl;
    matmul_kernel<T><<< dimGrid, dimBlock , shared_mem_size >>>(A, B, C, N, block_dim);
    cudaCheckError();
}


__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim)
{
    //Int version.
    matmul<int>(A , B , C , n, block_dim);
}
__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n, unsigned int block_dim)
{
    //Float version.
    matmul<float>(A , B , C , n, block_dim);
}
__host__ void matmul_3(const double *A, const double *B, double *C, unsigned int n, unsigned int block_dim)
{
    //Double version.
    matmul<double>(A , B , C , n, block_dim);
} 


