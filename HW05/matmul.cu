#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include "profile.cuh"



template <typename T>
__global__ void matmul_kernel(const T *A, const T *B, T *C, unsigned int N, unsigned int BLOCK_SIZE)
{
    extern __shared__ char smem[];
    T* sData = reinterpret_cast<T *>(smem);

    T* As = sData;
    T* Bs = sData + BLOCK_SIZE * BLOCK_SIZE;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin= N * BLOCK_SIZE * by;
    int aEnd= aBegin + N - 1; 
    int aStep= BLOCK_SIZE;

    int bBegin= BLOCK_SIZE * bx;
    int bStep= BLOCK_SIZE * N;

    T Csub = 0;

    for (int a = aBegin, b = bBegin;a <= aEnd;a += aStep, b += bStep) 
    {
        As[ty * BLOCK_SIZE + tx] = A[a + N * ty + tx];
        Bs[ty * BLOCK_SIZE + tx] = B[b + N * ty + tx];

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[ty * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx];

        __syncthreads();
    }

    int c = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + N * ty + tx] = Csub;
}

template <typename T>
__host__ void matmul(const T *A, const T *B, T *C, unsigned int N,  unsigned int block_dim)
{
    // Launch simple kernel on GPU with 2 block and 8 threads.
    float f_N = N;
    float f_block_dim = block_dim;
    size_t grid_size = ceil(f_N / f_block_dim);
    dim3 dimBlock( block_dim, block_dim );
    dim3 dimGrid( grid_size , grid_size );
    size_t shared_mem_size = 2 * block_dim * block_dim * sizeof(T); // two mini matrices of size block_dim.
    //std::cout << block_dim << "X" << grid_size << " SM: " << shared_mem_size << " " << N << std::endl;
    matmul_kernel<T><<< dimGrid, dimBlock , shared_mem_size >>>(A, B, C, N, block_dim);
    cudaCheckError();
}


__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim)
{
    matmul<int>(A , B , C , n, block_dim);
}
__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n, unsigned int block_dim)
{
    matmul<float>(A , B , C , n, block_dim);
}
__host__ void matmul_3(const double *A, const double *B, double *C, unsigned int n, unsigned int block_dim)
{
    matmul<double>(A , B , C , n, block_dim);
} 


