#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "reduce.cuh"
#include "profile.cuh"

#include <iostream>
#include <string> 
#include <random>

using std::cout;
using std::endl;


void printX(float *val, int N)
{
    std::cout << "Array : " << std::endl;
    for(int i = 0; i < N ; i++)
    {
        std::cout << val[i] << " ";
    }
    std::cout << std::endl;
}

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        cout << "Usage ./task1 N threads per block" << endl;
        return 0;
    }
    //Prepare variable for calculations
    size_t N = std::stoi(argv[1]);
    size_t threads_per_block = std::stoi(argv[2]);

    float *input, *output;     
    float *d_input, *d_output;
    size_t input_size = N * sizeof(float);

    double f_N = N;
    double array_size_per_block = 2 * threads_per_block;
    size_t out_N = ceil( f_N / array_size_per_block );
    size_t output_size = out_N * sizeof(float); 

    
    // Generate Random Values for kernel
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source()); 
    std::uniform_real_distribution<float> dist(-1.0,1.0);
    
    // Allocate space for device and host array a
    hipMalloc((void **)&d_input, input_size);
    cudaCheckError();
    hipMalloc((void **)&d_output, output_size);
    cudaCheckError();
    
    input = (float *)malloc(input_size);
    output = (float *)malloc(output_size);
    
    // Fill a, b, c array on host
    for(size_t i = 0; i < N ; i++)
    {
        input[i] = i + 1;
        if(i < out_N)
            output[i] = 0;
    }

    //Copy data from host to device
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    cudaCheckError();
    hipMemcpy(d_output, output, output_size, hipMemcpyHostToDevice);
    cudaCheckError();

    float time_taken = 0;
    {
        UnitGPUTime g;
        reduce(&d_input, &d_output, N, threads_per_block);
        time_taken = g.getTime();
    }
    // Copy result back to host
    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    cudaCheckError();
    hipMemcpy(input, d_input, output_size, hipMemcpyDeviceToHost);
    cudaCheckError();
    //Print last element and time taken.
    // printX(input, out_N);
    // printX(output, out_N);
    
    cout << std::log2(N) << "," << time_taken << endl;

    // Cleanup
    free(input);
    free(output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}