#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "reduce.cuh"
#include "profile.cuh"

#include <iostream>
#include <string> 
#include <random>

using std::cout;
using std::endl;


void printX(float *val, int N)
{
    std::cout << "Array : " << std::endl;
    for(int i = 0; i < N ; i++)
    {
        std::cout << val[i] << " ";
    }
    std::cout << std::endl;
}

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        cout << "Usage ./task1 N threads per block" << endl;
        return 0;
    }
    //Prepare variable for calculations
    size_t N = std::stoi(argv[1]);
    size_t threads_per_block = std::stoi(argv[2]);

    float *input, *output;     
    float *d_input, *d_output;
    size_t size = N * sizeof(float);
    
    // Generate Random Values for kernel
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source()); 
    std::uniform_real_distribution<float> dist(-1.0,1.0);
    
    // Allocate space for device and host array a
    hipMalloc((void **)&d_input, size);
    cudaCheckError();
    hipMalloc((void **)&d_output, size);
    cudaCheckError();
    
    input = (float *)malloc(size);
    output = (float *)malloc(size);
    
    // Fill a, b, c array on host
    for(size_t i = 0; i < N ; i++)
    {
        input[i] = i + 1;
        output[i] = 0;
    }

    //Copy data from host to device
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    cudaCheckError();
    hipMemcpy(d_output, output, size, hipMemcpyHostToDevice);
    cudaCheckError();

    float time_taken = 0;
    {
        UnitGPUTime g;
        reduce(&d_input, &d_output, N, threads_per_block);
        time_taken = g.getTime();
    }
    // Copy result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    cudaCheckError();
    hipMemcpy(input, d_input, size, hipMemcpyDeviceToHost);
    cudaCheckError();
    //Print last element and time taken.
    // printX(input, N);
    // printX(output, N);
    
    cout << std::log2(N) << "," << time_taken << endl;

    // Cleanup
    free(input);
    free(output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}