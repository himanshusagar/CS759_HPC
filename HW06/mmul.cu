#include <iostream>
#include <chrono>
#include "mmul.h"
#include "profile.cuh"

static const char * __cublasGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";

        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";

        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
    }

    return "<unknown>";
}

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int N)
{   
    const float alpha = 1.0f;
    const float beta  = 1.0f;

    hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, 
        &alpha, A, N, B, N, &beta, C, N);
    if(status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("hipblasSgemm error %s" , __cublasGetErrorEnum(status));
    }
    hipDeviceSynchronize();
    cudaCheckError();
}